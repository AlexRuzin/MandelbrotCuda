#include "hip/hip_runtime.h"
#include "cudaMandelbrot.h"

#include "cuda_occupancy.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define CUDA_MANDELBROT_INTERATIONS 32

float elapsedTime = 0;

using namespace cuda;

__constant__ rgbaPixel pixel_colour[16] =
{
	{ 66,  30,  15 },
	{ 25,   7,  26 },
	{ 9,   1,  47 },
	{ 4,   4,  73 },
	{ 0,   7, 100 },
	{ 12,  44, 138 },
	{ 24,  82, 177 },
	{ 57, 125, 209 },
	{ 134, 181, 229 },
	{ 211, 236, 248 },
	{ 241, 233, 191 },
	{ 248, 201,  95 },
	{ 255, 170,   0 },
	{ 204, 128,   0 },
	{ 153,  87,   0 },
	{ 106,  52,   3 }
};

__global__ void mandelbrot_kernel(rgbaPixel* image,
	int32_t width, int32_t height,
	double scale,
	double cx, double cy);

template<class T, typename... A>
error_t cudaKernel::launch_kernel(T& kernel, dim3 work, A&&... args)
{
	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);

	int threadBlocks;
	if (props.major == 2)
	{
		threadBlocks = 8;
	}
	else if (props.major == 3)
	{
		threadBlocks = 16;
	}
	else
	{
		threadBlocks = 32;
	}

	threadBlocks = 8;

	int blockSize;
	std::uint32_t minGridSize;
	hipOccupancyMaxPotentialBlockSize((int*)&minGridSize, &blockSize, kernel, 0, 0);
	
	int maxActiveBlocks = 0;
	do
	{
		hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, kernel, blockSize, 0);

		if (blockSize < props.warpSize || maxActiveBlocks >= threadBlocks)
		{
			break;
		}

		blockSize -= props.warpSize;
	} while (true);	

	int blockSizeDimX, blockSizeDimY;
	blockSizeDimX = blockSizeDimY = (int)pow(2, ceil(log(sqrt(blockSize)) / log(2)));

	while (blockSizeDimX * blockSizeDimY > blockSize)
	{
		blockSizeDimY--;
	}

	dim3 block(blockSizeDimX, blockSizeDimY);
	dim3 grid((work.x + block.x - 1) / block.x, (work.y + block.y - 1) / block.y);
	grid.x = grid.x > minGridSize ? grid.x : minGridSize;
	grid.y = grid.y > minGridSize ? grid.y : minGridSize;

#ifdef CUDA_DEBUG_OUT
	float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize);

	std::cout << "Grid of size " << grid.x * grid.y << std::endl;
	std::cout << "Launched blocks of size " << blockSize << std::endl;
	std::cout << "Theoretical occupancy " << occupancy * 100.0f << "%" << std::endl;
#endif //CUDA_DEBUG_OUT

	hipEvent_t start;
	hipEventCreate(&start);

	hipEvent_t stop;
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	kernel << < grid, block >> > (std::forward<A>(args)...);

	hipGetLastError();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipProfilerStop();

	return 0;
}

error_t cudaKernel::generate_mandelbrot(void)
{
	rgbaPixel *cudaBuffer;

	hipMalloc((void**)&cudaBuffer, pixelBufferRawSize);
	hipMemset(cudaBuffer, 0, pixelBufferRawSize);

	error_t err = launch_kernel(mandelbrot_kernel,
		dim3((int32_t)pixelLength, (int32_t)pixelHeight), cudaBuffer,
		(int32_t)pixelLength, (int32_t)pixelHeight, scale, offsetX, offsetY);
	if (err != 0) {
		return err;
	}

	hipMemcpy((void*)&this->pixelBuffer, cudaBuffer, pixelBufferRawSize, hipMemcpyDeviceToHost);
	hipFree(cudaBuffer);

	return 0;
}

__global__ void mandelbrot_kernel(rgbaPixel* image,
	int32_t width, int32_t height,
	double scale,
	double cx, double cy)
{
	const int i = threadIdx.y + blockIdx.y * blockDim.y;
	const int j = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= height || j >= width)
	{
		return;
	}

	const std::uint8_t max_iter = 255;
	const double y = (i - (height >> 1)) * scale + cy;
	const double x = (j - (width >> 1)) * scale + cx;

	double zx = hypot(x - 0.25, y);

	if (x < zx - 2.0 * zx * zx + 0.25 || (x + 1.0) * (x + 1.0) + y * y < 0.0625)
	{
		return;
	}

	std::uint8_t iter = 0;
	double zy, zx2, zy2;
	zx = zy = zx2 = zy2 = 0.0;

	do {
		zy = 2.0 * zx * zy + y;
		zx = zx2 - zy2 + x;
		zx2 = zx * zx;
		zy2 = zy * zy;
	} while (iter++ < max_iter && zx2 + zy2 < 4.0);

	if (iter > 0 && iter < max_iter)
	{
		const std::uint8_t colour_idx = iter % 16;

		image[i * width + j] = pixel_colour[colour_idx];
	}
}